#include "hip/hip_runtime.h"
#include <cmath>
#include "GSTrevise.hpp"
#include <stdio.h>
#include <chrono>


__global__ void cudaGenerateCurve(float* dCurveX, float* dCurveY, float area, float start, float interval, int size)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  float x = start + id * interval;
  if (id < size)
  {
    dCurveX[id] = x;
    dCurveY[id] = area / x;
  }
}

void GPUgenerateCurve(Node n, GST& gst, float* dCurveX, float* dCurveY, int blockSize = 512)
{
  auto& nodeData = gst.nodes[n];
  std::cout << "curve size = " << nodeData.shapeCurveX.size() << "\n";

  size_t arraySize = 1048576;
  int iBytes = arraySize * sizeof(float);

  int gridSize = (arraySize + blockSize - 1) / blockSize;

  dim3 block(blockSize);
  dim3 grid(gridSize);

  float maxX = sqrt(nodeData.area / nodeData.par1);
  float minX = sqrt(nodeData.area / nodeData.par2);

  float interval = (maxX - minX) / arraySize;

  auto startKernel = std::chrono::high_resolution_clock::now();
  cudaGenerateCurve<<<grid, block>>>(dCurveX, dCurveY, nodeData.area, minX, interval, arraySize);
  hipDeviceSynchronize();
  auto endKernel = std::chrono::high_resolution_clock::now();

  std::vector<float> hCurveX(arraySize);
  std::vector<float> hCurveY(arraySize);

  auto startMemcpy = std::chrono::high_resolution_clock::now();
  hipMemcpy(hCurveX.data(), dCurveX, arraySize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hCurveY.data(), dCurveY, arraySize * sizeof(float), hipMemcpyDeviceToHost);
  auto endMemcpy = std::chrono::high_resolution_clock::now();

  auto startMove = std::chrono::high_resolution_clock::now();
  nodeData.shapeCurveX = std::move(hCurveX);
  nodeData.shapeCurveY = std::move(hCurveY);
  auto endMove = std::chrono::high_resolution_clock::now();


  std::cout << "Kernel execution took: " 
            << std::chrono::duration_cast<std::chrono::nanoseconds>(endKernel - startKernel).count() << " ns\n";
  std::cout << "Memory copy took: " 
            << std::chrono::duration_cast<std::chrono::nanoseconds>(endMemcpy - startMemcpy).count() << " ns\n";
  std::cout << "Move operations took: " 
            << std::chrono::duration_cast<std::chrono::nanoseconds>(endMove - startMove).count() << " ns\n";
}


// int main()
// {
//   size_t arraySize = 512;
//   std::vector<std::vector<float>> shapeX(3, std::vector<float>(512, 0));
//   std::vector<std::vector<float>> shapeY(3, std::vector<float>(512, 0));
  
//   std::vector<Subcircuit> functionList(3);
//   functionList[0] = Subcircuit(10, false, true, 0.3, 3);
//   functionList[1] = Subcircuit(20, false, true, 0.3, 3);
//   functionList[2] = Subcircuit(30, false, true, 0.2, 5);

//   for (auto& i : functionList)
//   {
//     cpuGenerateCurve(i);
//   }
// }



void measureExecutionTime(const std::string& label, const std::function<void()>& func) {
    auto start = std::chrono::high_resolution_clock::now();

    func();
    auto end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    std::cout << label << " took " << duration << " ms\n";
}

int main() {
  GST gst = fakePartition();

  size_t arraySize = 1048576;
  int iBytes = arraySize * sizeof(float);

  float *dCurveX, *dCurveY;

  auto startMalloc = std::chrono::high_resolution_clock::now();
  hipMalloc((float**)&dCurveX, iBytes);
  hipMalloc((float**)&dCurveY, iBytes);
  auto endMalloc = std::chrono::high_resolution_clock::now();

  hipMemset(dCurveX, 0, iBytes);
  hipMemset(dCurveY, 0, iBytes);

  std::cout << "Memory allocation took: " 
            << std::chrono::duration_cast<std::chrono::nanoseconds>(endMalloc - startMalloc).count() << " ns\n";


  std::cout << "Measuring CPU function:\n";
  measureExecutionTime("CPU Function", [&]() {

  generatePoints(1, gst);
  });

  std::cout << "Measuring GPU function:\n";
  measureExecutionTime("GPU Function", [&]() {
    GPUgenerateCurve(2, gst, dCurveX, dCurveY);
  });
  std::cout << "Measuring GPU function:\n";
  measureExecutionTime("GPU Function", [&]() {
    GPUgenerateCurve(3, gst, dCurveX, dCurveY);
  });

  return 0;
}